#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define GROUP_SIZE 5

__global__ void mandelKernel(int *device_data, float lowerX, float lowerY, float stepX, float stepY, size_t pitch, int maxIterations){
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    
    // find base index for each thread group
    int thisX = (blockIdx.x * blockDim.x + threadIdx.x) * GROUP_SIZE;
    int thisY = (blockIdx.y * blockDim.y + threadIdx.y) * GROUP_SIZE;

    // process a group of pixels
    for(int i = 0; i < GROUP_SIZE; i++){
        for(int j = 0; j < GROUP_SIZE; j++){
            // find desire pixel in the thread group
            int localX = thisX + j;
            int localY = thisY + i;

            // initialize mandel variables
            float c_re = lowerX + localX * stepX;
            float c_im = lowerY + localY * stepY;
            float z_re = c_re, z_im = c_im;

            // pointer points to the pixel should be processed in this thread
            int* ptr = (int*) ((char*) device_data + localY * pitch) + localX;

            // by theorem in mandel, if |c| <= 0.25 then c belongs to M
            if(z_re * z_re + z_im * z_im <= 0.25f * 0.25f){
                *ptr = maxIterations;
                continue;
            }
            
            // mandel iteration
            int intensity;
            for(intensity = 0; intensity < maxIterations; intensity++){
                if(z_re * z_re + z_im * z_im > 4.f)
                    break;

                float new_re = z_re * z_re - z_im * z_im;
                float new_im = 2.f * z_re * z_im;
                z_re = c_re + new_re;
                z_im = c_im + new_im;
            }
            
            *ptr = intensity;
        }
    }
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE(float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations){
    // compute steps
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    // allocate memory
    int N = resX * resY;
    int *host_data;
    hipHostAlloc((void**) &host_data, N * sizeof(int), hipHostMallocMapped);

    int *device_data;
    size_t pitch;
    hipMallocPitch(&device_data, &pitch, resX * sizeof(int), resY);

    // launch kernel function
    dim3 threads_per_block(20, 20);
    dim3 num_blocks(resX / (threads_per_block.x * GROUP_SIZE), resY / (threads_per_block.y * GROUP_SIZE));
    mandelKernel<<<num_blocks, threads_per_block>>>(device_data, lowerX, lowerY, stepX, stepY, pitch, maxIterations);
    
    // wait for kernel function finish
    hipDeviceSynchronize();

    // output answers
    hipMemcpy2D(host_data, resX * sizeof(int), device_data, pitch, resX * sizeof(int), resY, hipMemcpyDeviceToHost);
    memcpy(img, host_data, N * sizeof(int));
    
    // free memory
    hipFree(device_data);
    hipHostFree(host_data);
}